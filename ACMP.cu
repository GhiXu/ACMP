#include "hip/hip_runtime.h"
#include "ACMP.h"

__device__  void sort_small(float *d, const int n)
{
    int j;
    for (int i = 1; i < n; i++) {
        float tmp = d[i];
        for (j = i; j >= 1 && tmp < d[j-1]; j--)
            d[j] = d[j-1];
        d[j] = tmp;
    }
}

__device__ void sort_small_weighted(float *d, float *w, int n)
{
    int j;
    for (int i = 1; i < n; i++) {
        float tmp = d[i];
        float tmp_w = w[i];
        for (j = i; j >= 1 && tmp < d[j - 1]; j--) {
            d[j] = d[j - 1];
            w[j] = w[j - 1];
        }
        d[j] = tmp;
        w[j] = tmp_w;
    }
}

__device__ int FindMinCostIndex(const float *costs, const int n)
{
    float min_cost = costs[0];
    int min_cost_idx = 0;
    for (int idx = 1; idx < n; ++idx) {
        if (costs[idx] <= min_cost) {
            min_cost = costs[idx];
            min_cost_idx = idx;
        }
    }
    return min_cost_idx;
}

__device__ int FindMaxCostIndex(const float *costs, const int n)
{
    float max_cost = costs[0];
    int max_cost_idx = 0;
    for (int idx = 1; idx < n; ++idx) {
        if (costs[idx] >= max_cost) {
            max_cost = costs[idx];
            max_cost_idx = idx;
        }
    }
    return max_cost_idx;
}

__device__  void setBit(unsigned int &input, const unsigned int n)
{
    input |= (unsigned int)(1 << n);
}

__device__  int isSet(unsigned int input, const unsigned int n)
{
    return (input >> n) & 1;
}

__device__ void Mat33DotVec3(const float mat[9], const float4 vec, float4 *result)
{
  result->x = mat[0] * vec.x + mat[1] * vec.y + mat[2] * vec.z;
  result->y = mat[3] * vec.x + mat[4] * vec.y + mat[5] * vec.z;
  result->z = mat[6] * vec.x + mat[7] * vec.y + mat[8] * vec.z;
}

__device__ float Vec3DotVec3(const float4 vec1, const float4 vec2)
{
    return vec1.x * vec2.x + vec1.y * vec2.y + vec1.z * vec2.z;
}

__device__ void NormalizeVec3 (float4 *vec)
{
    const float normSquared = vec->x * vec->x + vec->y * vec->y + vec->z * vec->z;
    const float inverse_sqrt = rsqrtf (normSquared);
    vec->x *= inverse_sqrt;
    vec->y *= inverse_sqrt;
    vec->z *= inverse_sqrt;
}

__device__ void TransformPDFToCDF(float* probs, const int num_probs)
{
    float prob_sum = 0.0f;
    for (int i = 0; i < num_probs; ++i) {
        prob_sum += probs[i];
    }
    const float inv_prob_sum = 1.0f / prob_sum;

    float cum_prob = 0.0f;
    for (int i = 0; i < num_probs; ++i) {
        const float prob = probs[i] * inv_prob_sum;
        cum_prob += prob;
        probs[i] = cum_prob;
    }
}

__device__ void Get3DPoint(const Camera camera, const int2 p, const float depth, float *X)
{
    X[0] = depth * (p.x - camera.K[2]) / camera.K[0];
    X[1] = depth * (p.y - camera.K[5]) / camera.K[4];
    X[2] = depth;
}

__device__ float4 GetViewDirection(const Camera camera, const int2 p, const float depth)
{
    float X[3];
    Get3DPoint(camera, p, depth, X);
    float norm = sqrt(X[0] * X[0] + X[1] * X[1] + X[2] * X[2]);

    float4 view_direction;
    view_direction.x = X[0] / norm;
    view_direction.y = X[1] / norm;
    view_direction.z =  X[2] / norm;
    view_direction.w = 0;
    return view_direction;
}

__device__ float GetDistance2Origin(const Camera camera, const int2 p, const float depth, const float4 normal)
{
    float X[3];
    Get3DPoint(camera, p, depth, X);
    return -(normal.x * X[0] + normal.y * X[1] + normal.z * X[2]);
}

__device__ float ComputeDepthfromPlaneHypothesis(const Camera camera, const float4 plane_hypothesis, const int2 p)
{
    return -plane_hypothesis.w * camera.K[0] / ((p.x - camera.K[2]) * plane_hypothesis.x + (camera.K[0] / camera.K[4]) * (p.y - camera.K[5]) * plane_hypothesis.y + camera.K[0] * plane_hypothesis.z);
}

__device__ float4 GenerateRandomNormal(const Camera camera, const int2 p, hiprandState *rand_state, const float depth)
{
    float4 normal;
    float q1 = 1.0f;
    float q2 = 1.0f;
    float s = 2.0f;
    while (s >= 1.0f) {
        q1 = 2.0f * hiprand_uniform(rand_state) -1.0f;
        q2 = 2.0f * hiprand_uniform(rand_state) - 1.0f;
        s = q1 * q1 + q2 * q2;
    }
    const float sq = sqrt(1.0f - s);
    normal.x = 2.0f * q1 * sq;
    normal.y = 2.0f * q2 * sq;
    normal.z = 1.0f - 2.0f * s;
    normal.w = 0;

    float4 view_direction = GetViewDirection(camera, p, depth);
    float dot_product = normal.x * view_direction.x + normal.y * view_direction.y + normal.z * view_direction.z;
    if (dot_product > 0.0f) {
        normal.x = -normal.x;
        normal.y = -normal.y;
        normal.z = - normal.z;
    }
    NormalizeVec3(&normal);
    return normal;
}

__device__ float4 GeneratePerturbedNormal(const Camera camera, const int2 p, const float4 normal, hiprandState *rand_state, const float perturbation)
{
    float4 view_direction = GetViewDirection(camera, p, 1.0f);

    const float a1 = (hiprand_uniform(rand_state) - 0.5f) * perturbation;
    const float a2 = (hiprand_uniform(rand_state) - 0.5f) * perturbation;
    const float a3 = (hiprand_uniform(rand_state) - 0.5f) * perturbation;

    const float sin_a1 = sin(a1);
    const float sin_a2 = sin(a2);
    const float sin_a3 = sin(a3);
    const float cos_a1 = cos(a1);
    const float cos_a2 = cos(a2);
    const float cos_a3 = cos(a3);

    float R[9];
    R[0] = cos_a2 * cos_a3;
    R[1] = cos_a3 * sin_a1 * sin_a2 - cos_a1 * sin_a3;
    R[2] = sin_a1 * sin_a3 + cos_a1 * cos_a3 * sin_a2;
    R[3] = cos_a2 * sin_a3;
    R[4] = cos_a1 * cos_a3 + sin_a1 * sin_a2 * sin_a3;
    R[5] = cos_a1 * sin_a2 * sin_a3 - cos_a3 * sin_a1;
    R[6] = -sin_a2;
    R[7] = cos_a2 * sin_a1;
    R[8] = cos_a1 * cos_a2;

    float4 normal_perturbed;
    Mat33DotVec3(R, normal, &normal_perturbed);

    if (Vec3DotVec3(normal_perturbed, view_direction) >= 0.0f) {
        normal_perturbed = normal;
    }

    NormalizeVec3(&normal_perturbed);
    return normal_perturbed;
}

__device__ float4 GenerateRandomPlaneHypothesis(const Camera camera, const int2 p, hiprandState *rand_state, const float depth_min, const float depth_max)
{
    float depth = hiprand_uniform(rand_state) * (depth_max - depth_min) + depth_min;
    float4 plane_hypothesis = GenerateRandomNormal(camera, p, rand_state, depth);
    plane_hypothesis.w = GetDistance2Origin(camera, p, depth, plane_hypothesis);
    return plane_hypothesis;
}

__device__ float4 GeneratePertubedPlaneHypothesis(const Camera camera, const int2 p, hiprandState *rand_state, const float perturbation, const float4 plane_hypothesis_now, const float depth_now, const float depth_min, const float depth_max)
{
    float depth_perturbed = depth_now;

    float dist_perturbed = plane_hypothesis_now.w;
    const float dist_min_perturbed = (1 - perturbation) * dist_perturbed;
    const float dist_max_perturbed = (1 + perturbation) * dist_perturbed;
    float4 plane_hypothesis_temp = plane_hypothesis_now;
    do {
        dist_perturbed = hiprand_uniform(rand_state) * (dist_max_perturbed - dist_min_perturbed) + dist_min_perturbed;
        plane_hypothesis_temp.w = dist_perturbed;
        depth_perturbed = ComputeDepthfromPlaneHypothesis(camera, plane_hypothesis_temp, p);
    } while (depth_perturbed < depth_min && depth_perturbed > depth_max);

    float4 plane_hypothesis = GeneratePerturbedNormal(camera, p, plane_hypothesis_now, rand_state, perturbation * M_PI);
    plane_hypothesis.w = dist_perturbed;
    return plane_hypothesis;
}

__device__ void ComputeHomography(const Camera ref_camera, const Camera src_camera, const float4 plane_hypothesis, float *H)
{
    float ref_C[3];
    float src_C[3];
    ref_C[0] = -(ref_camera.R[0] * ref_camera.t[0] + ref_camera.R[3] * ref_camera.t[1] + ref_camera.R[6] * ref_camera.t[2]);
    ref_C[1] = -(ref_camera.R[1] * ref_camera.t[0] + ref_camera.R[4] * ref_camera.t[1] + ref_camera.R[7] * ref_camera.t[2]);
    ref_C[2] = -(ref_camera.R[2] * ref_camera.t[0] + ref_camera.R[5] * ref_camera.t[1] + ref_camera.R[8] * ref_camera.t[2]);
    src_C[0] = -(src_camera.R[0] * src_camera.t[0] + src_camera.R[3] * src_camera.t[1] + src_camera.R[6] * src_camera.t[2]);
    src_C[1] = -(src_camera.R[1] * src_camera.t[0] + src_camera.R[4] * src_camera.t[1] + src_camera.R[7] * src_camera.t[2]);
    src_C[2] = -(src_camera.R[2] * src_camera.t[0] + src_camera.R[5] * src_camera.t[1] + src_camera.R[8] * src_camera.t[2]);

    float R_relative[9];
    float C_relative[3];
    float t_relative[3];
    R_relative[0] = src_camera.R[0] * ref_camera.R[0] + src_camera.R[1] * ref_camera.R[1] + src_camera.R[2] *ref_camera.R[2];
    R_relative[1] = src_camera.R[0] * ref_camera.R[3] + src_camera.R[1] * ref_camera.R[4] + src_camera.R[2] *ref_camera.R[5];
    R_relative[2] = src_camera.R[0] * ref_camera.R[6] + src_camera.R[1] * ref_camera.R[7] + src_camera.R[2] *ref_camera.R[8];
    R_relative[3] = src_camera.R[3] * ref_camera.R[0] + src_camera.R[4] * ref_camera.R[1] + src_camera.R[5] *ref_camera.R[2];
    R_relative[4] = src_camera.R[3] * ref_camera.R[3] + src_camera.R[4] * ref_camera.R[4] + src_camera.R[5] *ref_camera.R[5];
    R_relative[5] = src_camera.R[3] * ref_camera.R[6] + src_camera.R[4] * ref_camera.R[7] + src_camera.R[5] *ref_camera.R[8];
    R_relative[6] = src_camera.R[6] * ref_camera.R[0] + src_camera.R[7] * ref_camera.R[1] + src_camera.R[8] *ref_camera.R[2];
    R_relative[7] = src_camera.R[6] * ref_camera.R[3] + src_camera.R[7] * ref_camera.R[4] + src_camera.R[8] *ref_camera.R[5];
    R_relative[8] = src_camera.R[6] * ref_camera.R[6] + src_camera.R[7] * ref_camera.R[7] + src_camera.R[8] *ref_camera.R[8];
    C_relative[0] = (ref_C[0] - src_C[0]);
    C_relative[1] = (ref_C[1] - src_C[1]);
    C_relative[2] = (ref_C[2] - src_C[2]);
    t_relative[0] = src_camera.R[0] * C_relative[0] + src_camera.R[1] * C_relative[1] + src_camera.R[2] * C_relative[2];
    t_relative[1] = src_camera.R[3] * C_relative[0] + src_camera.R[4] * C_relative[1] + src_camera.R[5] * C_relative[2];
    t_relative[2] = src_camera.R[6] * C_relative[0] + src_camera.R[7] * C_relative[1] + src_camera.R[8] * C_relative[2];

    H[0] = R_relative[0] - t_relative[0] * plane_hypothesis.x / plane_hypothesis.w;
    H[1] = R_relative[1] - t_relative[0] * plane_hypothesis.y / plane_hypothesis.w;
    H[2] = R_relative[2] - t_relative[0] * plane_hypothesis.z / plane_hypothesis.w;
    H[3] = R_relative[3] - t_relative[1] * plane_hypothesis.x / plane_hypothesis.w;
    H[4] = R_relative[4] - t_relative[1] * plane_hypothesis.y / plane_hypothesis.w;
    H[5] = R_relative[5] - t_relative[1] * plane_hypothesis.z / plane_hypothesis.w;
    H[6] = R_relative[6] - t_relative[2] * plane_hypothesis.x / plane_hypothesis.w;
    H[7] = R_relative[7] - t_relative[2] * plane_hypothesis.y / plane_hypothesis.w;
    H[8] = R_relative[8] - t_relative[2] * plane_hypothesis.z / plane_hypothesis.w;

    float tmp[9];
    tmp[0] = H[0] / ref_camera.K[0];
    tmp[1] = H[1] / ref_camera.K[4];
    tmp[2] = -H[0] * ref_camera.K[2] / ref_camera.K[0] - H[1] * ref_camera.K[5] / ref_camera.K[4] + H[2];
    tmp[3] = H[3] / ref_camera.K[0];
    tmp[4] = H[4] / ref_camera.K[4];
    tmp[5] = -H[3] * ref_camera.K[2] / ref_camera.K[0] - H[4] * ref_camera.K[5] / ref_camera.K[4] + H[5];
    tmp[6] = H[6] / ref_camera.K[0];
    tmp[7] = H[7] / ref_camera.K[4];
    tmp[8] = -H[6] * ref_camera.K[2] / ref_camera.K[0] - H[7] * ref_camera.K[5] / ref_camera.K[4] + H[8];

    H[0] = src_camera.K[0] * tmp[0] + src_camera.K[2] * tmp[6];
    H[1] = src_camera.K[0] * tmp[1] + src_camera.K[2] * tmp[7];
    H[2] = src_camera.K[0] * tmp[2] + src_camera.K[2] * tmp[8];
    H[3] = src_camera.K[4] * tmp[3] + src_camera.K[5] * tmp[6];
    H[4] = src_camera.K[4] * tmp[4] + src_camera.K[5] * tmp[7];
    H[5] = src_camera.K[4] * tmp[5] + src_camera.K[5] * tmp[8];
    H[6] = src_camera.K[8] * tmp[6];
    H[7] = src_camera.K[8] * tmp[7];
    H[8] = src_camera.K[8] * tmp[8];
}

__device__ float2 ComputeCorrespondingPoint(const float *H, const int2 p)
{
    float3 pt;
    pt.x = H[0] * p.x + H[1] * p.y + H[2];
    pt.y = H[3] * p.x + H[4] * p.y + H[5];
    pt.z = H[6] * p.x + H[7] * p.y + H[8];
    return make_float2(pt.x / pt.z, pt.y / pt.z);
}

__device__ float4 TransformNormal(const Camera camera, float4 plane_hypothesis)
{
    float4 transformed_normal;
    transformed_normal.x = camera.R[0] * plane_hypothesis.x + camera.R[3] * plane_hypothesis.y + camera.R[6] * plane_hypothesis.z;
    transformed_normal.y = camera.R[1] * plane_hypothesis.x + camera.R[4] * plane_hypothesis.y + camera.R[7] * plane_hypothesis.z;
    transformed_normal.z = camera.R[2] * plane_hypothesis.x + camera.R[5] * plane_hypothesis.y + camera.R[8] * plane_hypothesis.z;
    transformed_normal.w = plane_hypothesis.w;
    return transformed_normal;
}

__device__ float4 TransformNormal2RefCam(const Camera camera, float4 plane_hypothesis)
{
    float4 transformed_normal;
    transformed_normal.x = camera.R[0] * plane_hypothesis.x + camera.R[1] * plane_hypothesis.y + camera.R[2] * plane_hypothesis.z;
    transformed_normal.y = camera.R[3] * plane_hypothesis.x + camera.R[4] * plane_hypothesis.y + camera.R[5] * plane_hypothesis.z;
    transformed_normal.z = camera.R[6] * plane_hypothesis.x + camera.R[7] * plane_hypothesis.y + camera.R[8] * plane_hypothesis.z;
    transformed_normal.w = plane_hypothesis.w;
    return transformed_normal;
}

__device__ float ComputeBilateralWeight(const float x_dist, const float y_dist, const float pix, const float center_pix, const float sigma_spatial, const float sigma_color)
{
    const float spatial_dist = sqrt(x_dist * x_dist + y_dist * y_dist);
    const float color_dist = fabs(pix - center_pix);
    return exp(-spatial_dist / (2.0f * sigma_spatial* sigma_spatial) - color_dist / (2.0f * sigma_color * sigma_color));
}

__device__ float ComputeBilateralNCC(const hipTextureObject_t ref_image, const Camera ref_camera, const hipTextureObject_t src_image, const Camera src_camera, const int2 p, const float4 plane_hypothesis, const PatchMatchParams params)
{
    const float cost_max = 2.0f;
    int radius = params.patch_size / 2;

    float H[9];
    ComputeHomography(ref_camera, src_camera, plane_hypothesis, H);
    float2 pt = ComputeCorrespondingPoint(H, p);
    if (pt.x >= src_camera.width || pt.x < 0.0f || pt.y >= src_camera.height || pt.y < 0.0f) {
        return cost_max;
    }

    float cost = 0.0f;
    {
        float sum_ref = 0.0f;
        float sum_ref_ref = 0.0f;
        float sum_src = 0.0f;
        float sum_src_src = 0.0f;
        float sum_ref_src = 0.0f;
        float bilateral_weight_sum = 0.0f;
        const float ref_center_pix = tex2D<float>(ref_image, p.x + 0.5f, p.y + 0.5f);

        for (int i = -radius; i < radius + 1; i += params.radius_increment) {
            float sum_ref_row = 0.0f;
            float sum_src_row = 0.0f;
            float sum_ref_ref_row = 0.0f;
            float sum_src_src_row = 0.0f;
            float sum_ref_src_row = 0.0f;
            float bilateral_weight_sum_row = 0.0f;

            for (int j = -radius; j < radius + 1; j += params.radius_increment) {
                const int2 ref_pt = make_int2(p.x + i, p.y + j);
                const float ref_pix = tex2D<float>(ref_image, ref_pt.x + 0.5f, ref_pt.y + 0.5f);
                float2 src_pt = ComputeCorrespondingPoint(H, ref_pt);
                const float src_pix = tex2D<float>(src_image, src_pt.x + 0.5f, src_pt.y + 0.5f);

                float weight = ComputeBilateralWeight(i, j, ref_pix, ref_center_pix, params.sigma_spatial, params.sigma_color);

                sum_ref_row += weight * ref_pix;
                sum_ref_ref_row += weight * ref_pix * ref_pix;
                sum_src_row += weight * src_pix;
                sum_src_src_row += weight * src_pix * src_pix;
                sum_ref_src_row += weight * ref_pix * src_pix;
                bilateral_weight_sum_row += weight;
            }

            sum_ref += sum_ref_row;
            sum_ref_ref += sum_ref_ref_row;
            sum_src += sum_src_row;
            sum_src_src += sum_src_src_row;
            sum_ref_src += sum_ref_src_row;
            bilateral_weight_sum += bilateral_weight_sum_row;
        }
        const float inv_bilateral_weight_sum = 1.0f / bilateral_weight_sum;
        sum_ref *= inv_bilateral_weight_sum;
        sum_ref_ref *= inv_bilateral_weight_sum;
        sum_src *= inv_bilateral_weight_sum;
        sum_src_src *= inv_bilateral_weight_sum;
        sum_ref_src *= inv_bilateral_weight_sum;

        const float var_ref = sum_ref_ref - sum_ref * sum_ref;
        const float var_src = sum_src_src - sum_src * sum_src;

        const float kMinVar = 1e-5f;
        if (var_ref < kMinVar || var_src < kMinVar) {
            return cost = cost_max;
        } else {
            const float covar_src_ref = sum_ref_src - sum_ref * sum_src;
            const float var_ref_src = sqrt(var_ref * var_src);
            return cost = max(0.0f, min(cost_max, 1.0f - covar_src_ref / var_ref_src));
        }
    }
}

__device__ float ComputeMultiViewInitialCostandSelectedViews(const hipTextureObject_t *images, const Camera *cameras, const int2 p, const float4 plane_hypothesis, unsigned int *selected_views, const PatchMatchParams params)
{
    float cost_max = 2.0f;
    float cost_vector[32] = {2.0f};
    float cost_vector_copy[32] = {2.0f};
    int cost_count = 0;
    int num_valid_views = 0;

    for (int i = 1; i < params.num_images; ++i) {
        float c = ComputeBilateralNCC(images[0], cameras[0], images[i], cameras[i], p, plane_hypothesis, params);
        cost_vector[i - 1] = c;
        cost_vector_copy[i - 1] = c;
        cost_count++;
        if (c < cost_max) {
            num_valid_views++;
        }
    }

    sort_small(cost_vector, cost_count);
    *selected_views = 0;

    int top_k = min(num_valid_views, params.top_k);
    if (top_k > 0) {
        float cost = 0.0f;
        for (int i = 0; i < top_k; ++i) {
            cost += cost_vector[i];
        }
        float cost_threshold = cost_vector[top_k - 1];
        for (int i = 0; i < params.num_images - 1; ++i) {
            if (cost_vector_copy[i] <= cost_threshold) {
                setBit(*selected_views, i);
            }
        }
        return cost / top_k;
    } else {
        return cost_max;
    }
}

__device__ void ComputeMultiViewCostVector(const hipTextureObject_t *images, const Camera *cameras, const int2 p, const float4 plane_hypothesis, float *cost_vector, const PatchMatchParams params)
{
    for (int i = 1; i < params.num_images; ++i) {
        cost_vector[i - 1] = ComputeBilateralNCC(images[0], cameras[0], images[i], cameras[i], p, plane_hypothesis, params);
    }
}

__device__ float3 Get3DPointonWorld_cu(const float x, const float y, const float depth, const Camera camera)
{
    float3 pointX;
    float3 tmpX;
    // Reprojection
    pointX.x = depth * (x - camera.K[2]) / camera.K[0];
    pointX.y = depth * (y - camera.K[5]) / camera.K[4];
    pointX.z = depth;

    // Rotation
    tmpX.x = camera.R[0] * pointX.x + camera.R[3] * pointX.y + camera.R[6] * pointX.z;
    tmpX.y = camera.R[1] * pointX.x + camera.R[4] * pointX.y + camera.R[7] * pointX.z;
    tmpX.z = camera.R[2] * pointX.x + camera.R[5] * pointX.y + camera.R[8] * pointX.z;

    // Transformation
    float3 C;
    C.x = -(camera.R[0] * camera.t[0] + camera.R[3] * camera.t[1] + camera.R[6] * camera.t[2]);
    C.y = -(camera.R[1] * camera.t[0] + camera.R[4] * camera.t[1] + camera.R[7] * camera.t[2]);
    C.z = -(camera.R[2] * camera.t[0] + camera.R[5] * camera.t[1] + camera.R[8] * camera.t[2]);
    pointX.x = tmpX.x + C.x;
    pointX.y = tmpX.y + C.y;
    pointX.z = tmpX.z + C.z;

    return pointX;
}

__device__ void ProjectonCamera_cu(const float3 PointX, const Camera camera, float2 &point, float &depth)
{
    float3 tmp;
    tmp.x = camera.R[0] * PointX.x + camera.R[1] * PointX.y + camera.R[2] * PointX.z + camera.t[0];
    tmp.y = camera.R[3] * PointX.x + camera.R[4] * PointX.y + camera.R[5] * PointX.z + camera.t[1];
    tmp.z = camera.R[6] * PointX.x + camera.R[7] * PointX.y + camera.R[8] * PointX.z + camera.t[2];

    depth = camera.K[6] * tmp.x + camera.K[7] * tmp.y + camera.K[8] * tmp.z;
    point.x = (camera.K[0] * tmp.x + camera.K[1] * tmp.y + camera.K[2] * tmp.z) / depth;
    point.y = (camera.K[3] * tmp.x + camera.K[4] * tmp.y + camera.K[5] * tmp.z) / depth;
}

__device__ float ComputeGeomConsistencyCost(const hipTextureObject_t depth_image, const Camera ref_camera, const Camera src_camera, const float4 plane_hypothesis, const int2 p)
{
    const float max_cost = 5.0f;

    float depth = ComputeDepthfromPlaneHypothesis(ref_camera, plane_hypothesis, p);
    float3 forward_point = Get3DPointonWorld_cu(p.x, p.y, depth, ref_camera);

    float2 src_pt;
    float src_d;
    ProjectonCamera_cu(forward_point, src_camera, src_pt, src_d);
    const float src_depth = tex2D<float>(depth_image,  (int)src_pt.x + 0.5f, (int)src_pt.y + 0.5f);

    if (src_depth == 0.0f) {
        return max_cost;
    }

    float3 src_3D_pt = Get3DPointonWorld_cu(src_pt.x, src_pt.y, src_depth, src_camera);

    float2 backward_point;
    float ref_d;
    ProjectonCamera_cu(src_3D_pt, ref_camera, backward_point, ref_d);

    const float diff_col = p.x - backward_point.x;
    const float diff_row = p.y - backward_point.y;
    return min(max_cost, sqrt(diff_col * diff_col + diff_row * diff_row));
}

__global__ void RandomInitialization(cudaTextureObjects *texture_objects, Camera *cameras, float4 *plane_hypotheses, float *costs, hiprandState *rand_states, unsigned int *selected_views, float4 *prior_planes, unsigned int *plane_masks, const PatchMatchParams params)
{
    const int2 p = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
    int width = cameras[0].width;
    int height = cameras[0].height;

    if (p.x >= width || p.y >= height) {
        return;
    }

    const int center = p.y * width + p.x;
    hiprand_init(clock64(), p.y, p.x, &rand_states[center]);

    if (params.geom_consistency) {
        float4 plane_hypothesis = plane_hypotheses[center];
        plane_hypothesis = TransformNormal2RefCam(cameras[0], plane_hypothesis);
        float depth = plane_hypothesis.w;
        plane_hypothesis.w = GetDistance2Origin(cameras[0], p, depth, plane_hypothesis);
        plane_hypotheses[center] = plane_hypothesis;
        costs[center] = ComputeMultiViewInitialCostandSelectedViews(texture_objects[0].images, cameras, p, plane_hypotheses[center], &selected_views[center], params);
    }
    else if (params.planar_prior) {
        if (plane_masks[center] > 0 && costs[center] >= 0.1f) {
            float perturbation = 0.02f;

            float4 plane_hypothesis = prior_planes[center];
            float depth_perturbed = plane_hypothesis.w;
            const float depth_min_perturbed = (1 - 3 * perturbation) * depth_perturbed;
            const float depth_max_perturbed = (1 + 3 * perturbation) * depth_perturbed;
            depth_perturbed = hiprand_uniform(&rand_states[center]) * (depth_max_perturbed - depth_min_perturbed) + depth_min_perturbed;
            float4 plane_hypothesis_perturbed = GeneratePerturbedNormal(cameras[0], p, plane_hypothesis, &rand_states[center], 3 * perturbation * M_PI);
            plane_hypothesis_perturbed.w = depth_perturbed;
            plane_hypotheses[center] = plane_hypothesis_perturbed;
            costs[center] = ComputeMultiViewInitialCostandSelectedViews(texture_objects[0].images, cameras, p, plane_hypotheses[center], &selected_views[center], params);
        }
        else {
            float4 plane_hypothesis = plane_hypotheses[center];
            float depth = plane_hypothesis.w;
            plane_hypothesis.w = GetDistance2Origin(cameras[0], p, depth, plane_hypothesis);
            plane_hypotheses[center] = plane_hypothesis;
            costs[center] = ComputeMultiViewInitialCostandSelectedViews(texture_objects[0].images, cameras, p, plane_hypotheses[center], &selected_views[center], params);
        }
    }
    else {
        plane_hypotheses[center] = GenerateRandomPlaneHypothesis(cameras[0], p, &rand_states[center], params.depth_min, params.depth_max);
        costs[center] = ComputeMultiViewInitialCostandSelectedViews(texture_objects[0].images, cameras, p, plane_hypotheses[center], &selected_views[center], params);
    }
}

__device__ void PlaneHypothesisRefinement(const hipTextureObject_t *images, const hipTextureObject_t *depth_images, const Camera *cameras, float4 *plane_hypothesis, float *depth, float *cost, hiprandState *rand_state, const float *view_weights, const float weight_norm, float4 *prior_planes, unsigned int *plane_masks, float *restricted_cost, const int2 p, const PatchMatchParams params)
{
    float perturbation = 0.02f;
    const int center = p.y * cameras[0].width + p.x;

    float gamma = 0.5f;
    float depth_sigma = (params.depth_max - params.depth_min) / 64.0f;
    float two_depth_sigma_squared = 2 * depth_sigma * depth_sigma;
    float angle_sigma = M_PI * (5.0f / 180.0f);
    float two_angle_sigma_squared = 2 * angle_sigma * angle_sigma;
    float beta = 0.18f;
    float depth_prior = 0.0f;

    float depth_rand;
    float4 plane_hypothesis_rand;
    if (params.planar_prior && plane_masks[center] > 0) {
        depth_prior = ComputeDepthfromPlaneHypothesis(cameras[0], prior_planes[center], p);
        depth_rand = hiprand_uniform(rand_state) * 6 * depth_sigma + (depth_prior - 3 * depth_sigma);
        plane_hypothesis_rand = GeneratePerturbedNormal(cameras[0], p, prior_planes[center], rand_state, angle_sigma);
    }
    else {
        depth_rand = hiprand_uniform(rand_state) * (params.depth_max - params.depth_min) + params.depth_min;
        plane_hypothesis_rand = GenerateRandomNormal(cameras[0], p, rand_state, *depth);
    }
    float depth_perturbed = *depth;
    const float depth_min_perturbed = (1 - perturbation) * depth_perturbed;
    const float depth_max_perturbed = (1 + perturbation) * depth_perturbed;
    do {
        depth_perturbed = hiprand_uniform(rand_state) * (depth_max_perturbed - depth_min_perturbed) + depth_min_perturbed;
    } while (depth_perturbed < params.depth_min && depth_perturbed > params.depth_max);
    float4 plane_hypothesis_perturbed = GeneratePerturbedNormal(cameras[0], p, *plane_hypothesis, rand_state, perturbation * M_PI); // GeneratePertubedPlaneHypothesis(cameras[0], p, rand_state, perturbation, *plane_hypothesis, *depth, params.depth_min, params.depth_max);

    const int num_planes = 5;
    float depths[num_planes] = {depth_rand, *depth, depth_rand, *depth, depth_perturbed};
    float4 normals[num_planes] = {*plane_hypothesis, plane_hypothesis_rand, plane_hypothesis_rand, plane_hypothesis_perturbed, *plane_hypothesis};

    for (int i = 0; i < num_planes; ++i) {
        float cost_vector[32] = {2.0f};
        float4 temp_plane_hypothesis = normals[i];
        temp_plane_hypothesis.w = GetDistance2Origin(cameras[0], p, depths[i], temp_plane_hypothesis); // dists[i];
        ComputeMultiViewCostVector(images, cameras, p, temp_plane_hypothesis, cost_vector, params);

        float temp_cost = 0.0f;
        for (int j = 0; j < params.num_images - 1; ++j) {
            if (view_weights[j] > 0) {
                if (params.geom_consistency) {
                    temp_cost += view_weights[j] * (cost_vector[j] + 0.1f * ComputeGeomConsistencyCost(depth_images[j+1], cameras[0], cameras[j+1], temp_plane_hypothesis, p));
                }
                else {
                    temp_cost += view_weights[j] * cost_vector[j];
                }
            }
        }
        temp_cost /= weight_norm;

        float depth_before = ComputeDepthfromPlaneHypothesis(cameras[0], temp_plane_hypothesis, p);
        if (params.planar_prior && plane_masks[center] > 0) {
            float depth_diff = depths[i] - depth_prior;
            float angle_cos = Vec3DotVec3(prior_planes[center], temp_plane_hypothesis);
            float angle_diff = acos(angle_cos);
            float prior = gamma + exp(- depth_diff * depth_diff / two_depth_sigma_squared) * exp(- angle_diff * angle_diff / two_angle_sigma_squared);
            float restricted_temp_cost = exp(-temp_cost * temp_cost / beta) * prior;
            if (depth_before >= params.depth_min && depth_before <= params.depth_max && restricted_temp_cost > *restricted_cost) {
                *depth = depth_before;
                *plane_hypothesis = temp_plane_hypothesis;
                *cost = temp_cost;
                *restricted_cost = restricted_temp_cost;
            }
        }
        else {
            if (depth_before >= params.depth_min && depth_before <= params.depth_max && temp_cost < *cost) {
                *depth = depth_before;
                *plane_hypothesis = temp_plane_hypothesis;
                *cost = temp_cost;
            }
        }
    }
}

__device__ void CheckerboardPropagation(const hipTextureObject_t *images, const hipTextureObject_t *depths, const Camera *cameras, float4 *plane_hypotheses, float *costs, hiprandState *rand_states, unsigned int *selected_views, float4 *prior_planes, unsigned int *plane_masks, const int2 p, const PatchMatchParams params, const int iter)
{
    int width = cameras[0].width;
    int height = cameras[0].height;
    if (p.x >= width || p.y >= height) {
        return;
    }

    const int center = p.y * width + p.x;
    int left_near = center - 1;
    int left_far = center - 3;
    int right_near = center + 1;
    int right_far = center + 3;
    int up_near = center - width;
    int up_far = center - 3 * width;
    int down_near = center + width;
    int down_far = center + 3 * width;

    // Adaptive Checkerboard Sampling
    float cost_array[8][32] = {2.0f};
    // 0 -- up_near, 1 -- up_far, 2 -- down_near, 3 -- down_far, 4 -- left_near, 5 -- left_far, 6 -- right_near, 7 -- right_far
    bool flag[8] = {false};
    int num_valid_pixels = 0;

    float costMin;
    int costMinPoint;

    // up_far
    if (p.y > 2) {
        flag[1] = true;
        num_valid_pixels++;
        costMin = costs[up_far];
        costMinPoint = up_far;
        for (int i = 1; i < 11; ++i) {
            if (p.y > 2 + 2 * i) {
                int pointTemp = up_far - 2 * i * width;
                if (costs[pointTemp] < costMin) {
                    costMin = costs[pointTemp];
                    costMinPoint = pointTemp;
                }
            }
        }
        up_far = costMinPoint;
        ComputeMultiViewCostVector(images, cameras, p, plane_hypotheses[up_far], cost_array[1], params);
    }

    // dwon_far
    if (p.y < height - 3) {
        flag[3] = true;
        num_valid_pixels++;
        costMin = costs[down_far];
        costMinPoint = down_far;
        for (int i = 1; i < 11; ++i) {
            if (p.y < height - 3 - 2 * i) {
                int pointTemp = down_far + 2 * i * width;
                if (costs[pointTemp] < costMin) {
                    costMin = costs[pointTemp];
                    costMinPoint = pointTemp;
                }
            }
        }
        down_far = costMinPoint;
        ComputeMultiViewCostVector(images, cameras, p, plane_hypotheses[down_far], cost_array[3], params);
    }

    // left_far
    if (p.x > 2) {
        flag[5] = true;
        num_valid_pixels++;
        costMin = costs[left_far];
        costMinPoint = left_far;
        for (int i = 1; i < 11; ++i) {
            if (p.x > 2 + 2 * i) {
                int pointTemp = left_far - 2 * i;
                if (costs[pointTemp] < costMin) {
                    costMin = costs[pointTemp];
                    costMinPoint = pointTemp;
                }
            }
        }
        left_far = costMinPoint;
        ComputeMultiViewCostVector(images, cameras, p, plane_hypotheses[left_far], cost_array[5], params);
    }

    // right_far
    if (p.x < width - 3) {
        flag[7] = true;
        num_valid_pixels++;
        costMin = costs[right_far];
        costMinPoint = right_far;
        for (int i = 1; i < 11; ++i) {
            if (p.x < width - 3 - 2 * i) {
                int pointTemp = right_far + 2 * i;
                if (costMin < costs[pointTemp]) {
                    costMin = costs[pointTemp];
                    costMinPoint = pointTemp;
                }
            }
        }
        right_far = costMinPoint;
        ComputeMultiViewCostVector(images, cameras, p, plane_hypotheses[right_far], cost_array[7], params);
    }

    // up_near
    if (p.y > 0) {
        flag[0] = true;
        num_valid_pixels++;
        costMin = costs[up_near];
        costMinPoint = up_near;
        for (int i = 0; i < 3; ++i) {
            if (p.y > 1 + i && p.x > i) {
                int pointTemp = up_near - (1 + i) * width - i;
                if (costs[pointTemp] < costMin) {
                    costMin = costs[pointTemp];
                    costMinPoint = pointTemp;
                }
            }
            if (p.y > 1 + i && p.x < width - 1 - i) {
                int pointTemp = up_near - (1 + i) * width + i;
                if (costs[pointTemp] < costMin) {
                    costMin = costs[pointTemp];
                    costMinPoint = pointTemp;
                }
            }
        }
        up_near = costMinPoint;
        ComputeMultiViewCostVector(images, cameras, p, plane_hypotheses[up_near], cost_array[0], params);
    }

    // down_near
    if (p.y < height - 1) {
        flag[2] = true;
        num_valid_pixels++;
        costMin = costs[down_near];
        costMinPoint = down_near;
        for (int i = 0; i < 3; ++i) {
            if (p.y < height - 2 - i && p.x > i) {
                int pointTemp = down_near + (1 + i) * width - i;
                if (costs[pointTemp] < costMin) {
                    costMin = costs[pointTemp];
                    costMinPoint = pointTemp;
                }
            }
            if (p.y < height - 2 - i && p.x < width - 1 - i) {
                int pointTemp = down_near + (1 + i) * width + i;
                if (costs[pointTemp] < costMin) {
                    costMin = costs[pointTemp];
                    costMinPoint = pointTemp;
                }
            }
        }
        down_near = costMinPoint;
        ComputeMultiViewCostVector(images, cameras, p, plane_hypotheses[down_near], cost_array[2], params);
    }

    // left_near
    if (p.x > 0) {
        flag[4] = true;
        num_valid_pixels++;
        costMin = costs[left_near];
        costMinPoint = left_near;
        for (int i = 0; i < 3; ++i) {
            if (p.x > 1 + i && p.y > i) {
                int pointTemp = left_near - (1 + i) - i * width;
                if (costs[pointTemp] < costMin) {
                    costMin = costs[pointTemp];
                    costMinPoint = pointTemp;
                }
            }
            if (p.x > 1 + i && p.y < height - 1 - i) {
                int pointTemp = left_near - (1 + i) + i * width;
                if (costs[pointTemp] < costMin) {
                    costMin = costs[pointTemp];
                    costMinPoint = pointTemp;
                }
            }
        }
        left_near = costMinPoint;
        ComputeMultiViewCostVector(images, cameras, p, plane_hypotheses[left_near], cost_array[4], params);
    }

    // right_near
    if (p.x < width - 1) {
        flag[6] = true;
        num_valid_pixels++;
        costMin = costs[right_near];
        costMinPoint = right_near;
        for (int i = 0; i < 3; ++i) {
            if (p.x < width - 2 - i && p.y > i) {
                int pointTemp = right_near + (1 + i) - i * width;
                if (costs[pointTemp] < costMin) {
                    costMin = costs[pointTemp];
                    costMinPoint = pointTemp;
                }
            }
            if (p.x < width - 2 - i && p.y < height - 1- i) {
                int pointTemp = right_near + (1 + i) + i * width;
                if (costs[pointTemp] < costMin) {
                    costMin = costs[pointTemp];
                    costMinPoint = pointTemp;
                }
            }
        }
        right_near = costMinPoint;
        ComputeMultiViewCostVector(images, cameras, p, plane_hypotheses[right_near], cost_array[6], params);
    }
    const int positions[8] = {up_near, up_far, down_near, down_far, left_near, left_far, right_near, right_far};

    // Multi-hypothesis Joint View Selection
    float view_weights[32] = {0.0f};
    float view_selection_priors[32] = {0.0f};
    int neighbor_positions[4] = {center - width, center + width, center - 1, center + 1};
    for (int i = 0; i < 4; ++i) {
        if (flag[2 * i]) {
            for (int j = 0; j < params.num_images - 1; ++j) {
                if (isSet(selected_views[neighbor_positions[i]], j) == 1) {
                    view_selection_priors[j] += 0.9f;
                } else {
                    view_selection_priors[j] += 0.1f;
                }
            }
        }
    }

    float sampling_probs[32] = {0.0f};
    float cost_threshold = 0.8 * expf((iter) * (iter) / (-90.0f));
    for (int i = 0; i < params.num_images - 1; i++) {
        float count = 0;
        int count_false = 0;
        float tmpw = 0;
        for (int j = 0; j < 8; j++) {
            if (cost_array[j][i] < cost_threshold) {
                tmpw += expf(cost_array[j][i] * cost_array[j][i] / (-0.18f));
                count++;
            }
            if (cost_array[j][i] > 1.2f) {
                count_false++;
            }
        }
        if (count > 2 && count_false < 3) {
            sampling_probs[i] = tmpw / count;
        }
        else if (count_false < 3) {
            sampling_probs[i] = expf(cost_threshold * cost_threshold / (-0.32f));
        }
        sampling_probs[i] = sampling_probs[i] * view_selection_priors[i];
    }

    TransformPDFToCDF(sampling_probs, params.num_images - 1);
    for (int sample = 0; sample < 15; ++sample) {
        const float rand_prob = hiprand_uniform(&rand_states[center]) - FLT_EPSILON;

        for (int image_id = 0; image_id < params.num_images - 1; ++image_id) {
            const float prob = sampling_probs[image_id];
            if (prob > rand_prob) {
                view_weights[image_id] += 1.0f;
                break;
            }
        }
    }

    unsigned int temp_selected_views = 0;
    int num_selected_view = 0;
    float weight_norm = 0;
    for (int i = 0; i < params.num_images - 1; ++i) {
        if (view_weights[i] > 0) {
            setBit(temp_selected_views, i);
            weight_norm += view_weights[i];
            num_selected_view++;
        }
    }

    float final_costs[8] = {0.0f};
    for (int i = 0; i < 8; ++i) {
        for (int j = 0; j < params.num_images - 1; ++j) {
            if (view_weights[j] > 0) {
                if (params.geom_consistency) {
                    if (flag[i]) {
                        final_costs[i] += view_weights[j] * (cost_array[i][j] + 0.1f * ComputeGeomConsistencyCost(depths[j+1], cameras[0], cameras[j+1], plane_hypotheses[positions[i]], p));
                    }
                    else {
                        final_costs[i] += view_weights[j] * (cost_array[i][j] + 0.1f * 5.0f);
                    }
                }
                else {
                    final_costs[i] += view_weights[j] * cost_array[i][j];
                }
            }
        }
        final_costs[i] /= weight_norm;
    }

    const int min_cost_idx = FindMinCostIndex(final_costs, 8);

    float cost_vector_now[32] = {2.0f};
    ComputeMultiViewCostVector(images, cameras, p, plane_hypotheses[center], cost_vector_now, params);
    float cost_now = 0.0f;
    for (int i = 0; i < params.num_images - 1; ++i) {
        if (params.geom_consistency) {
            cost_now += view_weights[i] * (cost_vector_now[i] + 0.1f * ComputeGeomConsistencyCost(depths[i+1], cameras[0], cameras[i+1], plane_hypotheses[center], p));
        }
        else {
            cost_now += view_weights[i] * cost_vector_now[i];
        }
    }
    cost_now /= weight_norm;
    costs[center] = cost_now;
    float depth_now = ComputeDepthfromPlaneHypothesis(cameras[0], plane_hypotheses[center], p);

    float restricted_cost = 0.0f;
    if (params.planar_prior) {
        float restricted_final_costs[8] = {0.0f};
        float gamma = 0.5f;
        float depth_sigma = (params.depth_max - params.depth_min) / 64.0f;
        float two_depth_sigma_squared = 2 * depth_sigma * depth_sigma;
        float angle_sigma = M_PI * (5.0f / 180.0f);
        float two_angle_sigma_squared = 2 * angle_sigma * angle_sigma;
        float depth_prior = ComputeDepthfromPlaneHypothesis(cameras[0], prior_planes[center], p);
        float beta = 0.18f;

        if (plane_masks[center] > 0) {
            for (int i = 0; i < 8; i++) {
                if (flag[i]) {
                    float depth_now = ComputeDepthfromPlaneHypothesis(cameras[0], plane_hypotheses[positions[i]], p);
                    float depth_diff = depth_now - depth_prior;
                    float angle_cos = Vec3DotVec3(prior_planes[center], plane_hypotheses[positions[i]]);
                    float angle_diff = acos(angle_cos);
                    float prior = gamma + exp(- depth_diff * depth_diff / two_depth_sigma_squared) * exp(- angle_diff * angle_diff / two_angle_sigma_squared);
                    restricted_final_costs[i] = exp(-final_costs[i] * final_costs[i] / beta) * prior;
                }
            }
            const int max_cost_idx = FindMaxCostIndex(restricted_final_costs, 8);

            float restricted_cost_now = 0.0f;
            float depth_now = ComputeDepthfromPlaneHypothesis(cameras[0], plane_hypotheses[center], p);
            float depth_diff = depth_now - depth_prior;
            float angle_cos = Vec3DotVec3(prior_planes[center], plane_hypotheses[center]);
            float angle_diff = acos(angle_cos);
            float prior = gamma + exp(- depth_diff * depth_diff / two_depth_sigma_squared) * exp(- angle_diff * angle_diff / two_angle_sigma_squared);
            restricted_cost_now = exp(-cost_now * cost_now / beta) * prior;

            if (flag[max_cost_idx]) {
                float depth_before = ComputeDepthfromPlaneHypothesis(cameras[0], plane_hypotheses[positions[max_cost_idx]], p);

                if (depth_before >= params.depth_min && depth_before <= params.depth_max && restricted_final_costs[max_cost_idx] > restricted_cost_now) {
                    depth_now   = depth_before;
                    plane_hypotheses[center] = plane_hypotheses[positions[max_cost_idx]];
                    costs[center] = final_costs[max_cost_idx];
                    restricted_cost = restricted_final_costs[max_cost_idx];
                    selected_views[center] = temp_selected_views;
                }
            }
        }
        else if (flag[min_cost_idx]) {
            float depth_before = ComputeDepthfromPlaneHypothesis(cameras[0], plane_hypotheses[positions[min_cost_idx]], p);

            if (depth_before >= params.depth_min && depth_before <= params.depth_max && final_costs[min_cost_idx] < cost_now) {
                depth_now = depth_before;
                plane_hypotheses[center] = plane_hypotheses[positions[min_cost_idx]];
                costs[center] = final_costs[min_cost_idx];
            }
        }
    }

    if (!params.planar_prior && flag[min_cost_idx]) {
        float depth_before = ComputeDepthfromPlaneHypothesis(cameras[0], plane_hypotheses[positions[min_cost_idx]], p);

        if (depth_before >= params.depth_min && depth_before <= params.depth_max && final_costs[min_cost_idx] < cost_now) {
            depth_now = depth_before;
            plane_hypotheses[center] = plane_hypotheses[positions[min_cost_idx]];
            costs[center] = final_costs[min_cost_idx];
            selected_views[center] = temp_selected_views;
        }
    }

    PlaneHypothesisRefinement(images, depths, cameras, &plane_hypotheses[center], &depth_now, &costs[center], &rand_states[center], view_weights, weight_norm, prior_planes, plane_masks, &restricted_cost, p, params);
}

__global__ void BlackPixelUpdate(cudaTextureObjects *texture_objects, cudaTextureObjects *texture_depths, Camera *cameras, float4 *plane_hypotheses, float *costs, hiprandState *rand_states, unsigned int *selected_views, float4 *prior_planes, unsigned int *plane_masks, const PatchMatchParams params, const int iter)
{
    int2 p = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
    if (threadIdx.x % 2 == 0) {
        p.y = p.y * 2;
    } else {
        p.y = p.y * 2 + 1;
    }

    CheckerboardPropagation(texture_objects[0].images, texture_depths[0].images, cameras, plane_hypotheses, costs, rand_states, selected_views, prior_planes, plane_masks, p, params, iter);
}

__global__ void RedPixelUpdate(cudaTextureObjects *texture_objects, cudaTextureObjects *texture_depths, Camera *cameras, float4 *plane_hypotheses, float *costs, hiprandState *rand_states, unsigned int *selected_views, float4 *prior_planes, unsigned int *plane_masks, const PatchMatchParams params, const int iter)
{
    int2 p = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
    if (threadIdx.x % 2 == 0) {
        p.y = p.y * 2 + 1;
    } else {
        p.y = p.y * 2;
    }

    CheckerboardPropagation(texture_objects[0].images, texture_depths[0].images, cameras, plane_hypotheses, costs, rand_states, selected_views, prior_planes, plane_masks, p, params, iter);
}

__global__ void GetDepthandNormal(Camera *cameras, float4 *plane_hypotheses, const PatchMatchParams params)
{
    const int2 p = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
    const int width = cameras[0].width;
    const int height = cameras[0].height;

    if (p.x >= width || p.y >= height) {
        return;
    }

    const int center = p.y * width + p.x;
    plane_hypotheses[center].w = ComputeDepthfromPlaneHypothesis(cameras[0], plane_hypotheses[center], p);
    plane_hypotheses[center] = TransformNormal(cameras[0], plane_hypotheses[center]);
}

__device__ void CheckerboardFilter(const Camera *cameras, float4 *plane_hypotheses, float *costs, const int2 p)
{
    int width = cameras[0].width;
    int height = cameras[0].height;
    if (p.x >= width || p.y >= height) {
        return;
    }

    const int center = p.y * width + p.x;

    float filter[21];
    int index = 0;

    filter[index++] = plane_hypotheses[center].w;

    // Left
    const int left = center - 1;
    const int leftleft = center - 3;

    // Up
    const int up = center - width;
    const int upup = center - 3 * width;

    // Down
    const int down = center + width;
    const int downdown = center + 3 * width;

    // Right
    const int right = center + 1;
    const int rightright = center + 3;

    if (costs[center] < 0.001f) {
        return;
    }

    if (p.y>0) {
        filter[index++] = plane_hypotheses[up].w;
    }
    if (p.y>2) {
        filter[index++] = plane_hypotheses[upup].w;
    }
    if (p.y>4) {
        filter[index++] = plane_hypotheses[upup-width*2].w;
    }
    if (p.y<height-1) {
        filter[index++] = plane_hypotheses[down].w;
    }
    if (p.y<height-3) {
        filter[index++] = plane_hypotheses[downdown].w;
    }
    if (p.y<height-5) {
        filter[index++] = plane_hypotheses[downdown+width*2].w;
    }
    if (p.x>0) {
        filter[index++] = plane_hypotheses[left].w;
    }
    if (p.x>2) {
        filter[index++] = plane_hypotheses[leftleft].w;
    }
    if (p.x>4) {
        filter[index++] = plane_hypotheses[leftleft-2].w;
    }
    if (p.x<width-1) {
        filter[index++] = plane_hypotheses[right].w;
    }
    if (p.x<width-3) {
        filter[index++] = plane_hypotheses[rightright].w;
    }
    if (p.x<width-5) {
        filter[index++] = plane_hypotheses[rightright+2].w;
    }
    if (p.y>0 &&
        p.x<width-2) {
        filter[index++] = plane_hypotheses[up+2].w;
    }
    if (p.y< height-1 &&
        p.x<width-2) {
        filter[index++] = plane_hypotheses[down+2].w;
    }
    if (p.y>0 &&
        p.x>1)
    {
        filter[index++] = plane_hypotheses[up-2].w;
    }
    if (p.y<height-1 &&
        p.x>1) {
        filter[index++] = plane_hypotheses[down-2].w;
    }
    if (p.x>0 &&
        p.y>2)
    {
        filter[index++] = plane_hypotheses[left  - width*2].w;
    }
    if (p.x<width-1 &&
        p.y>2)
    {
        filter[index++] = plane_hypotheses[right - width*2].w;
    }
    if (p.x>0 &&
        p.y<height-2) {
        filter[index++] = plane_hypotheses[left  + width*2].w;
    }
    if (p.x<width-1 &&
        p.y<height-2) {
        filter[index++] = plane_hypotheses[right + width*2].w;
    }

    sort_small(filter,index);
    int median_index = index / 2;
    if (index % 2 == 0) {
        plane_hypotheses[center].w = (filter[median_index-1] + filter[median_index]) / 2;
    } else {
        plane_hypotheses[center].w = filter[median_index];
    }
}

__global__ void BlackPixelFilter(const Camera *cameras, float4 *plane_hypotheses, float *costs)
{
    int2 p = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
    if (threadIdx.x % 2 == 0) {
        p.y = p.y * 2;
    } else {
        p.y = p.y * 2 + 1;
    }

    CheckerboardFilter(cameras, plane_hypotheses, costs, p);
}

__global__ void RedPixelFilter(const Camera *cameras, float4 *plane_hypotheses, float *costs)
{
    int2 p = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
    if (threadIdx.x % 2 == 0) {
        p.y = p.y * 2 + 1;
    } else {
        p.y = p.y * 2;
    }

    CheckerboardFilter(cameras, plane_hypotheses, costs, p);
}

void ACMP::RunPatchMatch()
{
    const int width = cameras[0].width;
    const int height = cameras[0].height;
    // std::cout << width << " " << height << std::endl;

    int BLOCK_W = 32;
    int BLOCK_H = (BLOCK_W / 2);

    dim3 grid_size_randinit;
    grid_size_randinit.x = (width + 16 - 1) / 16;
    grid_size_randinit.y=(height + 16 - 1) / 16;
    grid_size_randinit.z = 1;
    dim3 block_size_randinit;
    block_size_randinit.x = 16;
    block_size_randinit.y = 16;
    block_size_randinit.z = 1;

    dim3 grid_size_checkerboard;
    grid_size_checkerboard.x = (width + BLOCK_W - 1) / BLOCK_W;
    grid_size_checkerboard.y= ( (height / 2) + BLOCK_H - 1) / BLOCK_H;
    grid_size_checkerboard.z = 1;
    dim3 block_size_checkerboard;
    block_size_checkerboard.x = BLOCK_W;
    block_size_checkerboard.y = BLOCK_H;
    block_size_checkerboard.z = 1;

    int max_iterations = params.max_iterations;

    RandomInitialization<<<grid_size_randinit, block_size_randinit>>>(texture_objects_cuda, cameras_cuda, plane_hypotheses_cuda, costs_cuda, rand_states_cuda, selected_views_cuda, prior_planes_cuda, plane_masks_cuda, params);
    CUDA_SAFE_CALL(hipDeviceSynchronize());

    for (int i = 0; i < max_iterations; ++i) {
        BlackPixelUpdate<<<grid_size_checkerboard, block_size_checkerboard>>>(texture_objects_cuda, texture_depths_cuda, cameras_cuda, plane_hypotheses_cuda, costs_cuda, rand_states_cuda, selected_views_cuda, prior_planes_cuda, plane_masks_cuda, params, i);
        CUDA_SAFE_CALL(hipDeviceSynchronize());
        RedPixelUpdate<<<grid_size_checkerboard, block_size_checkerboard>>>(texture_objects_cuda, texture_depths_cuda, cameras_cuda, plane_hypotheses_cuda, costs_cuda, rand_states_cuda, selected_views_cuda, prior_planes_cuda, plane_masks_cuda, params, i);
        CUDA_SAFE_CALL(hipDeviceSynchronize());
        printf("iteration: %d\n", i);
    }

    GetDepthandNormal<<<grid_size_randinit, block_size_randinit>>>(cameras_cuda, plane_hypotheses_cuda, params);
    CUDA_SAFE_CALL(hipDeviceSynchronize());

    BlackPixelFilter<<<grid_size_checkerboard, block_size_checkerboard>>>(cameras_cuda, plane_hypotheses_cuda, costs_cuda);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    RedPixelFilter<<<grid_size_checkerboard, block_size_checkerboard>>>(cameras_cuda, plane_hypotheses_cuda, costs_cuda);
    CUDA_SAFE_CALL(hipDeviceSynchronize());

    hipMemcpy(plane_hypotheses_host, plane_hypotheses_cuda, sizeof(float4) * width * height, hipMemcpyDeviceToHost);
    hipMemcpy(costs_host, costs_cuda, sizeof(float) * width * height, hipMemcpyDeviceToHost);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
}
